
#include <hip/hip_runtime.h>
/*******

Developed by Timothy Lardner and Jerzy Dziewierz,
Centre for Ultrasonic Engineering,
University of Strathclyde,
Copyright 2017,

Not to be used or copied without authorization from the authors,
This superseeds any other licence that may be bundled with this software.

For more information see:
"Inspection of complex components using 2D arrays and TFM"
"A design methodology for 2D sparse NDE arrays  using an efficient implementation of refracted - ray TFM"

http://www.cue.ac.uk

*******/


#define COEFFGEN_nTimePointsPerLine 33  // 17 points in a line for calculation, 16 for checking the polyfit
#define classicMinSearch_spread 1e-3
#define COEFF_SIZE 5
#define COEFFGEN_nTimePointsPerToFit 17
#define COEFFGEN_nTimePointsPerToCheck 16

struct SurfParam
{
    float x1, y1, z1, x2, y2, z2;    // position of source and target point
    float slow1, slow2;              // slowness of wave in medium 1 and 2
    float c0, c1;// c2, c3, c4, c5, c6, c7, c8, c9, c10, c11, c12, c13, c14; // for ny,y0,dy,nx,x0,dx
  //  float y0limit, y1limit;
    float *DataVector;                  // can be used as a pointer to a longer surface descriptor
    int DataVectorElementCount;         // can be used as a pointer to a longer surface descriptor
};

//static SurfParam *GlobalParam = new SurfParam();

__global__  void TestStruct1(float* output,SurfParam* parameters){
    int idx=threadIdx.x+blockDim.x*blockIdx.x;
    output[idx] = parameters->slow1;
}


__device__ __host__ void polyfit17x5(double output[5], double A[34]);
__device__ __host__ inline float PerItem_TOF(SurfParam& Params);
__device__ __host__ inline float PerItem_TOF_CF(float &x, float &y, SurfParam& Params);
__device__ __host__ inline float PerItem(float &x,float &y, SurfParam& Params) ;
__device__ __host__ inline float FlatZ_TOF(SurfParam& Params);
__device__ __host__ inline float FlatZ_TOF_CF(float &x, float &y, SurfParam& Params);
__device__ __host__ inline float FlatZ(float &x,float &y, SurfParam& Params) ;
__device__ __host__ inline void sortArray(float &fv1, float &fv2, float &fv3, float &vx1, float &vy1, float &vx2, float &vy2, float &vx3, float &vy3);
__device__ __host__ inline void swapArray(float &a1,float &a2, float &a3,float &b1,float &b2,float &b3);
__device__ __host__ float classicMinSearch(float (*CostFunction)(float& ,float& ,SurfParam&), SurfParam& Params,float &vx0,float &vy0);
__host__ __device__ inline void ind2sub(int *siz, int idx, int *sub);
__device__ __host__ inline double Square(double &x);
__device__ __host__ inline double Reciprocal( double &x);
__device__ __host__ inline double Power3(double &x);
__device__ __host__ inline double Power4(double &x);



__global__ void transform_tpoints_into_coeffs2GPU_kernel(
        int total_coefflines,
        int kernel_ProbeElementCount,
        int kernel_ny,
        const float* ZVector,
        const float* TBuffer,
        float* CoeffBuffer
        )
{ // per-coeffset kernel
    int idx_coeffline=blockIdx.x * blockDim.x + threadIdx.x;
    if (idx_coeffline>total_coefflines) {return;}


            // convert idx_coeffline to idx_tx and idx_zline using ind2sub:
        int size[4];
        int subs[4];
        size[3]=kernel_ProbeElementCount;
        size[2]=kernel_ny;
        size[1]=1;
        size[0]=1;
        ind2sub(size,idx_coeffline,subs);
        int idx_tx=subs[3];
        int idx_zline=subs[2];

        // calculate buffer offsets:
        int CoefflineOffset=idx_coeffline*COEFF_SIZE;
        int ZVector_offset=0;

        int TBuffer_offset=(idx_tx+idx_zline*kernel_ProbeElementCount)*COEFFGEN_nTimePointsPerLine;

        //idx_NaNBuffer=0+2*(idx_tx+persistent_ProbeElementCount*(idx_zline));// not used here
        // local buffers, double precision for polyfit 17x5
        double fitinput[34];
        double polycoeff5[5];

        // fill the local buffers
        // NOTE: format for 'fitinput' is float [x0,x1,...,x16,y0,y1,...,y16] - this comes compiled from Mathematica
        // put tbase first
        //if (persistent_verbosemode){mexPrintf("tbase:\n");};
        for (int idx_tbase=0;idx_tbase<COEFFGEN_nTimePointsPerToFit; idx_tbase++)
        {
            fitinput[idx_tbase]=(double)(ZVector[2*idx_tbase+ZVector_offset]); // !note, every 2nd z-sample is taken for fit source table
            //if (persistent_verbosemode){mexPrintf("fitinput[%d]=%e\n",idx_tbase,fitinput[idx_tbase]);};
        }
        // THIS loads points from TBuffer to temporary, double-precision buffer suitable for PolyFit
        // put every 2nd timepoint now
        for (int idx_tbase=0;idx_tbase<COEFFGEN_nTimePointsPerToFit; idx_tbase++)
        {
            fitinput[COEFFGEN_nTimePointsPerToFit+idx_tbase]=(double)(TBuffer[TBuffer_offset+2*idx_tbase]); // !note, every 2nd z-sample is taken for fit source table
        }
        // transform the fit input into coeff buffer using the Mathematica-generated code
        polyfit17x5(polycoeff5,fitinput);
        // load the items from polycoeff5 to CoeffBuffer
        CoeffBuffer[CoefflineOffset+COEFF_SIZE-1]=(float)polycoeff5[0];
        CoeffBuffer[CoefflineOffset+COEFF_SIZE-2]=(float)polycoeff5[1];
        CoeffBuffer[CoefflineOffset+COEFF_SIZE-3]=(float)polycoeff5[2];
        CoeffBuffer[CoefflineOffset+COEFF_SIZE-4]=(float)polycoeff5[3];
        CoeffBuffer[CoefflineOffset+COEFF_SIZE-5]=(float)polycoeff5[4];
}

__global__ void GenerateTimePoints(float* TimeBuffer,float* ZVector, float* ydim, float* ProbeLocation, int SurfID, int n_elem, int ny, int NoOfTimePoints,SurfParam* parameters){

    int idx=threadIdx.x+blockDim.x*blockIdx.x; // Only launch 1D block/grid
    if (idx>NoOfTimePoints){
        return; // If we launch more threads than we have timepoints (and we shouldn't), don't run
    }


    int subs[4];
    int size[4];
    size[3] = (int)COEFFGEN_nTimePointsPerLine;
    size[2] = (int)n_elem; // Number of elements in the probe
    size[1] = (int)ny; // Number of pixels in the y-direction
    size[0] = 1;

    ind2sub(size,idx,subs);
    int yline_idx=subs[1];
    int probeElementIdx=subs[2];
    int timePointOfLine=subs[3];



    float probe_y=ProbeLocation[1+3*probeElementIdx];  // Y-Point of the probe element
    float probe_z=ProbeLocation[2+3*probeElementIdx];  // Z-Point of the probe element
    float yy=ydim[yline_idx];                          // Y-Point that this kernel is calculating for
    float zz=ZVector[timePointOfLine];          // Z-Point that this kernel is calculating for

    float tof; // The final time of flight



    SurfParam localParam = *parameters; // Create a local copy of the parameters for each thread
    localParam.y1 = probe_y;
    localParam.z1=probe_z;
    localParam.y2=yy;
    localParam.z2=zz;

    if(SurfID==0){
        // No refraction takes place
        float dy=probe_y-yy;
        float dz=probe_z-zz;
        tof=sqrtf(dy*dy+dz*dz)*localParam.slow1; // We invert the speed once as multiplying is faster than dividing
    }
    else if(SurfID==1){
        // FlatZ refraction takes place
        tof=FlatZ_TOF(localParam);
    }
    else if(SurfID==2){
        // 2D-Surface refraction takes place
        tof=PerItem_TOF(localParam);
    }
    TimeBuffer[idx]=tof;
}

__global__ void TFM_coeff(float *dest, float *FMC, float *Elem, int n_elem, float fs, float *zdim, int nz, int sample_length, float time_start,float* Coeffs)
{
  int idx=threadIdx.x+blockDim.x*blockIdx.x;

  float tx_path;
  float rx_path;
  float time;
  float accumulator = 0;
  int sample_truncated;
  float sample_weight;
  int z_location = idx/nz;
  int y_location = idx%nz;
  int final_sample;
  float z_position = zdim[z_location];
  int tx_coeff_offset;
  int rx_coeff_offset;

  for(int i=0; i<n_elem; i++){

      tx_coeff_offset = 5*y_location*n_elem + 5*i;
      tx_path = Coeffs[tx_coeff_offset];
      tx_path = z_position*tx_path + Coeffs[tx_coeff_offset+1];
      tx_path = z_position*tx_path + Coeffs[tx_coeff_offset+2];
      tx_path = z_position*tx_path + Coeffs[tx_coeff_offset+3];
      tx_path = z_position*tx_path + Coeffs[tx_coeff_offset+4];
      for(int j=0; j<n_elem;j++){
         rx_coeff_offset = 5*y_location*n_elem + 5*j;
         rx_path = Coeffs[rx_coeff_offset];
         rx_path = z_position*rx_path + Coeffs[rx_coeff_offset+1];
         rx_path = z_position*rx_path + Coeffs[rx_coeff_offset+2];
         rx_path = z_position*rx_path + Coeffs[rx_coeff_offset+3];
         rx_path = z_position*rx_path + Coeffs[rx_coeff_offset+4];
         sample_truncated = floorf(fminf(fmaxf(((tx_path+rx_path)- time_start)*fs,0.0f),sample_length-2.0f));
         sample_weight = fminf(fmaxf(time*fs-sample_truncated,0.0f),1.0f);
         final_sample = (i*n_elem+j)*sample_length + sample_truncated;
         accumulator = accumulator + (1.0-sample_weight)*FMC[final_sample] + sample_weight*FMC[final_sample+1];
      }
  }
  dest[idx] = accumulator;
  //dest[idx] = 1;
  
}

__global__ void TFM(float *dest, float *FMC, float *Elem, int n_elem, float *ydim, float* zdim, float speed, int ny, int nz,float fs,int sample_length, float time_start)
{
  int idx=threadIdx.x+blockDim.x*blockIdx.x;
  float tx_location;
  float rx_location;
  float tx_depth;
  float rx_depth;
  float tx_path;
  float rx_path;
  float time;
  float accumulator = 0;
  int sample_truncated;
  float sample_weight;
  int z_location = idx/nz;
  int y_location = idx%nz;
  int final_sample;

  for(int i=0; i<n_elem; i++){
      tx_location = Elem[3*i+1];
      tx_depth = Elem[3*i+2];
      tx_path = sqrt((tx_location-ydim[y_location])*(tx_location-ydim[y_location]) + (tx_depth-zdim[z_location])*(tx_depth-zdim[z_location]));

      for(int j=0; j<n_elem;j++){
         rx_location = Elem[3*j+1];
         rx_depth = Elem[3*j+2];
         rx_path = sqrt((rx_location-ydim[y_location])*(rx_location-ydim[y_location]) + (rx_depth-zdim[z_location])*(rx_depth-zdim[z_location]));
         sample_truncated = floorf(fminf(fmaxf(((tx_path+rx_path) / speed - time_start)*fs,0.0f),sample_length-2.0f));
         sample_weight = fminf(fmaxf(time*fs-sample_truncated,0.0f),1.0f);
         final_sample = (i*n_elem+j)*sample_length + sample_truncated;
         accumulator = accumulator + (1.0-sample_weight)*FMC[final_sample] + sample_weight*FMC[final_sample+1];
      }
  }
  //dest[idx] = rx_path/speed;
  dest[idx] = accumulator;
}

__device__ __host__ inline float PerItem_TOF(SurfParam& Params)
{

    float x0_guess=0.5*(Params.x1+Params.x2);
    float y0_guess=0.5*(Params.y1+Params.y2);
    float tof = classicMinSearch(&PerItem_TOF_CF,Params,x0_guess,y0_guess);
    return tof;
}

__device__ __host__ inline float PerItem_TOF_CF(float &x, float &y, SurfParam& Params)
{
    float surfaceZ=PerItem(x,y,Params);
    float dx1=Params.x1-x;
    float dy1=Params.y1-y;
    float dz1=Params.z1-surfaceZ;
    float dx2=x-Params.x2;
    float dy2=y-Params.y2;
    float dz2=surfaceZ-Params.z2;
    float tof=sqrtf(dx1*dx1+dy1*dy1+dz1*dz1)*Params.slow1+sqrtf(dx2*dx2+dy2*dy2+dz2*dz2)*Params.slow2;
    return tof;
}

__device__ __host__ inline float PerItem(float &x,float &y, SurfParam& Params)
{
    float pickupPointerF=fmaxf(0,fminf((float)Params.DataVectorElementCount-1,((y-Params.c0)/Params.c1)));
    float pickupPointerBase=floorf(pickupPointerF);
    float pickupPointerRem=pickupPointerF-pickupPointerBase; // use this to blend samples
    int pickupPointerBaseI=(int)pickupPointerBase;
    float result = 0;
    result=(1-pickupPointerRem)*Params.DataVector[pickupPointerBaseI]+(pickupPointerRem)*Params.DataVector[pickupPointerBaseI+1];
    return result;
}

__device__ __host__ inline float FlatZ_TOF(SurfParam& Params)
{
    float x0_guess=0.5*(Params.x1+Params.x2);
    float y0_guess=0.5*(Params.y1+Params.y2);
    float tof = classicMinSearch(&FlatZ_TOF_CF,Params,x0_guess,y0_guess);
    return tof;
}

__device__ __host__ inline float FlatZ_TOF_CF(float &x, float &y, SurfParam& Params)
{
   float surfaceZ=FlatZ(x,y,Params);
   float dx1=Params.x1-x;
   float dy1=Params.y1-y;
   float dz1=Params.z1-surfaceZ;
   float dx2=x-Params.x2;
   float dy2=y-Params.y2;
   float dz2=surfaceZ-Params.z2;
   float tof=sqrtf(dx1*dx1+dy1*dy1+dz1*dz1)*Params.slow1+sqrtf(dx2*dx2+dy2*dy2+dz2*dz2)*Params.slow2;
   return tof;
}

__device__ __host__ inline float FlatZ(float &x,float &y, SurfParam& Params)
{
    return float(0);
}

__host__ __device__ inline void ind2sub(int *siz, int idx, int *sub)
{
int prod[4];
                prod[0] = siz[3]*siz[2]*siz[1];
                prod[1] = siz[3]*siz[2];
                prod[2] = siz[3];
                prod[3] = 1;
                sub[0] = (int)floor(    (float)idx / prod[0]                                                    );
                sub[1] = (int)floor(    (float)(        idx % prod[0]   )/prod[1]                               );
                sub[2] = (int)floor( (float)( ( idx % prod[0]   )%prod[1]       )  / prod[2]);
                sub[3] =                ( (     idx % prod[0]   )%prod[1]       )  % prod[2] ;
}

__device__ __host__ float classicMinSearch(float (*CostFunction)(float&, float& ,SurfParam&), SurfParam& Params,float &vx0, float &vy0)
{

    #define tolf 1e-10

//! Exit condition, tolerance on X,Y value
    #define tolx 1e-5

//! Simplex method parameter, controls the way the problem space is explored
    #define rho 1

//! Simplex method parameter, controls the way the problem space is explored
    #define chi 2

//! Simplex method parameter, controls the way the problem space is explored
    #define psi 0.5

//! Simplex method parameter, controls the way the problem space is explored
    #define sigma 0.5

// Maximum number of iterations for Nelder-Mead search
    #define ITERATION_LIMIT 10000


    int how = 0;

    float vx1 = vx0;
    float vy1 = vy0;
    float fv1 = (*CostFunction)(vx1,vy1,Params);

    float vx2 = vx0+classicMinSearch_spread;
    float vy2 = vy0;
    float fv2 = (*CostFunction)(vx2,vy2,Params);

    float vx3 = vx0;
    float vy3 = vy0+classicMinSearch_spread;
    float fv3 = (*CostFunction)(vx3,vy3,Params);

    sortArray(fv1,fv2,fv3,vx1,vy1,vx2,vy2,vx3,vy3);



    int loops = 1;

    while((abs(fv2-fv1) > tolf)|(abs(vx1-vx2)>tolx)|(abs(vy1-vy2)>tolx))
    {
        loops++;
        if(loops>ITERATION_LIMIT){
            return (float)fv1; // return best approximation
        }

        float xbar = (vx1 + vx2)/2;
        float ybar = (vy1 + vy2)/2;
        float xr = (1 + rho)*xbar - rho*vx3;
        float yr = (1 + rho)*ybar - rho*vy3;
        float fxr = (*CostFunction)(xr,yr,Params);

        if(fxr<fv1){
            float xe = (1 + rho*chi)*xbar - rho*chi*vx3;
            float ye = (1 + rho*chi)*ybar - rho*chi*vy3;
            float fxe = (*CostFunction)(xe,ye,Params);
            if (fxe < fxr){
                vx3 = xe;
                vy3 = ye;
                fv3 = fxe;
            }
            else{
                vx3 = xr;
                vy3 = yr;
                fv3 = fxr;}
        }
        else{
            if (fxr < fv2){
                vx3 = xr;
                vy3 = yr;
                fv3 = fxr;
            }
            else{
                if(fxr < fv3){
                    float xc = (1 + psi*rho)*xbar - psi*rho*vx3;
                    float yc = (1 + psi*rho)*ybar - psi*rho*vy3;
                    float fxc = (*CostFunction)(xc,yc,Params);
                    if(fxc <=fxr){
                        vx3 = xc;
                        vy3 = yc;
                        fv3 = fxc;
                        how = 0;
                    }
                    else{
                        how = 1;
                    }
                }
                else{
                    float xcc = (1-psi)*xbar + psi*vx3;
                    float ycc = (1-psi)*ybar + psi*vx3;
                    double fxcc = (*CostFunction)(xcc,ycc,Params);
                    if(fxcc<fv3){
                        vx3 = xcc;
                        vy3 = ycc;
                        fv3 = fxcc;
                        how = 0;
                    }
                    else{
                        how = 1;
                    }

                }
                if(how){
                    vx2 = vx1 + sigma*(vx2 - vx1);
                    vy2 = vy1 + sigma*(vy2 - vy1);
                    fv2 = (*CostFunction)(vx2,vy2,Params);

                    vx3 = vx1 + sigma*(vx3 - vx1);
                    vy3 = vy1 + sigma*(vy3 - vy1);
                    fv3 = (*CostFunction)(vx3,vy3,Params);
                }

            }

        }
    sortArray(fv1, fv2, fv3, vx1, vy1, vx2, vy2, vx3, vy3);
    }
    vx0=vx1;
    vy0=vy1;
    return fv1;
}

__device__ __host__ void sortArray(float &fv1, float &fv2, float &fv3, float &vx1, float &vy1, float &vx2, float &vy2, float &vx3, float &vy3)
{
        swapArray(fv2,vx2,vy2,fv3,vx3,vy3);
        swapArray(fv1,vx1,vy1,fv2,vx2,vy2);
        swapArray(fv2,vx2,vy2,fv3,vx3,vy3);
}

__device__ __host__ void swapArray(float &a1,float &a2, float &a3,float &b1,float &b2,float &b3)
{
        float tmp;
        if (a1 > b1)
        {
            tmp=b1; b1=a1; a1=tmp;
            tmp=b2; b2=a2; a2=tmp;
            tmp=b3; b3=a3; a3=tmp;
        }
}

__device__ __host__ void polyfit17x5(double output[5], double A[34])
{

// NOTE: These constants below are true constants, but #defines are not best for them because variables like R236 can be used elsewhere
// TODO: Rename generic names like R236 to routine-specific names to replace them with defined constants


double R342 = 17.;
double R193 = -1.;

double R34 = A[0];
double R35 = A[9];
double R36 = A[10];
double R37 = A[11];
double R38 = A[12];
double R39 = A[13];
double R40 = A[14];
double R41 = A[15];
double R42 = A[16];
double R43 = A[1];
double R44 = A[2];
double R45 = A[3];
double R46 = A[4];
double R47 = A[5];
double R48 = A[6];
double R49 = A[7];
double R50 = A[8];
double R51 = Square( A[0]);
double R52 = Square( A[9]);
double R53 = Square( A[10]);
double R54 = Square( A[11]);
double R55 = Square( A[12]);
double R56 = Square( A[13]);
double R57 = Square( A[14]);
double R58 = Square( A[15]);
double R59 = Square( A[16]);
double R60 = Square( A[1]);
double R61 = Square( A[2]);
double R62 = Square( A[3]);
double R63 = Square( A[4]);
double R64 = Square( A[5]);
double R65 = Square( A[6]);
double R66 = Square( A[7]);
double R67 = Square( A[8]);
double R68 = Square( R34);
double R69 = Square( R35);
double R70 = Square( R36);
double R71 = Square( R37);
double R72 = Square( R38);
double R73 = Square( R39);
double R74 = Square( R40);
double R75 = Square( R41);
double R76 = Square( R42);
double R77 = Square( R43);
double R78 = Square( R44);
double R79 = Square( R45);
double R80 = Square( R46);
double R81 = Square( R47);
double R82 = Square( R48);
double R83 = Square( R49);
double R84 = Square( R50);
double R85 = Power3( A[0]);
double R86 = Power3( A[9]);
double R87 = Power3( A[10]);
double R88 = Power3( A[11]);
double R89 = Power3( A[12]);
double R90 = Power3( A[13]);
double R91 = Power3( A[14]);
double R92 = Power3( A[15]);
double R93 = Power3( A[16]);
double R94 = Power3( A[1]);
double R95 = Power3( A[2]);
double R96 = Power3( A[3]);
double R97 = Power3( A[4]);
double R98 = Power3( A[5]);
double R99 = Power3( A[6]);
double R100 = Power3( A[7]);
double R101 = Power3( A[8]);
double R102 = A[0] * R34;
double R103 = A[9] * R35;
double R104 = A[10] * R36;
double R105 = A[11] * R37;
double R106 = A[12] * R38;
double R107 = A[13] * R39;
double R108 = A[14] * R40;
double R109 = A[15] * R41;
double R110 = A[16] * R42;
double R111 = A[1] * R43;
double R112 = A[2] * R44;
double R113 = A[3] * R45;
double R114 = A[4] * R46;
double R115 = A[5] * R47;
double R116 = A[6] * R48;
double R117 = A[7] * R49;
double R118 = A[8] * R50;
double R119 = R102 + R103 + R104 + R105 + R106 + R107 + R108 + R109 + R110 + R111 + R112 + R113 + R114 + R115 + R116 + R117 + R118;
double R120 = A[0] + A[9] + A[10] + A[11] + A[12] + A[13] + A[14] + A[15] + A[16] + A[1] + A[2] + A[3] + A[4] + A[5] + A[6] + A[7] + A[8];
double R121 = R85 + R86 + R87 + R88 + R89 + R90 + R91 + R92 + R93 + R94 + R95 + R96 + R97 + R98 + R99 + R100 + R101;
double R122 = R51 * R34;
double R123 = R52 * R35;
double R124 = R53 * R36;
double R125 = R54 * R37;
double R126 = R55 * R38;
double R127 = R56 * R39;
double R128 = R57 * R40;
double R129 = R58 * R41;
double R130 = R59 * R42;
double R131 = R60 * R43;
double R132 = R61 * R44;
double R133 = R62 * R45;
double R134 = R63 * R46;
double R135 = R64 * R47;
double R136 = R65 * R48;
double R137 = R66 * R49;
double R138 = R67 * R50;
double R139 = R122 + R123 + R124 + R125 + R126 + R127 + R128 + R129 + R130 + R131 + R132 + R133 + R134 + R135 + R136 + R137 + R138;
double R140 = R51 + R52 + R53 + R54 + R55 + R56 + R57 + R58 + R59 + R60 + R61 + R62 + R63 + R64 + R65 + R66 + R67;
double R141 = R85 * R34;
double R142 = R86 * R35;
double R143 = R87 * R36;
double R144 = R88 * R37;
double R145 = R89 * R38;
double R146 = R90 * R39;
double R147 = R91 * R40;
double R148 = R92 * R41;
double R149 = R93 * R42;
double R150 = R94 * R43;
double R151 = R95 * R44;
double R152 = R96 * R45;
double R153 = R97 * R46;
double R154 = R98 * R47;
double R155 = R99 * R48;
double R156 = R100 * R49;
double R157 = R101 * R50;
double R158 = R141 + R142 + R143 + R144 + R145 + R146 + R147 + R148 + R149 + R150 + R151 + R152 + R153 + R154 + R155 + R156 + R157;
double R159 = Power3( R34);
double R160 = Power3( R35);
double R161 = Power3( R36);
double R162 = Power3( R37);
double R163 = Power3( R38);
double R164 = Power3( R39);
double R165 = Power3( R40);
double R166 = Power3( R41);
double R167 = Power3( R42);
double R168 = Power3( R43);
double R169 = Power3( R44);
double R170 = Power3( R45);
double R171 = Power3( R46);
double R172 = Power3( R47);
double R173 = Power3( R48);
double R174 = Power3( R49);
double R175 = Power3( R50);
double R176 = Power4( A[0]);
double R177 = Power4( A[9]);
double R178 = Power4( A[10]);
double R179 = Power4( A[11]);
double R180 = Power4( A[12]);
double R181 = Power4( A[13]);
double R182 = Power4( A[14]);
double R183 = Power4( A[15]);
double R184 = Power4( A[16]);
double R185 = Power4( A[1]);
double R186 = Power4( A[2]);
double R187 = Power4( A[3]);
double R188 = Power4( A[4]);
double R189 = Power4( A[5]);
double R190 = Power4( A[6]);
double R191 = Power4( A[7]);
double R192 = Power4( A[8]);
double R194 = R193 * R140 * R119;
double R195 = R120 * R139;
double R196 = R194 + R195;
double R197 = R51 * R68;
double R198 = R52 * R69;
double R199 = R53 * R70;
double R200 = R54 * R71;
double R201 = R55 * R72;
double R202 = R56 * R73;
double R203 = R57 * R74;
double R204 = R58 * R75;
double R205 = R59 * R76;
double R206 = R60 * R77;
double R207 = R61 * R78;
double R208 = R62 * R79;
double R209 = R63 * R80;
double R210 = R64 * R81;
double R211 = R65 * R82;
double R212 = R66 * R83;
double R213 = R67 * R84;
double R214 = R197 + R198 + R199 + R200 + R201 + R202 + R203 + R204 + R205 + R206 + R207 + R208 + R209 + R210 + R211 + R212 + R213;
double R215 = A[0] * R68;
double R216 = A[9] * R69;
double R217 = A[10] * R70;
double R218 = A[11] * R71;
double R219 = A[12] * R72;
double R220 = A[13] * R73;
double R221 = A[14] * R74;
double R222 = A[15] * R75;
double R223 = A[16] * R76;
double R224 = A[1] * R77;
double R225 = A[2] * R78;
double R226 = A[3] * R79;
double R227 = A[4] * R80;
double R228 = A[5] * R81;
double R229 = A[6] * R82;
double R230 = A[7] * R83;
double R231 = A[8] * R84;
double R232 = R215 + R216 + R217 + R218 + R219 + R220 + R221 + R222 + R223 + R224 + R225 + R226 + R227 + R228 + R229 + R230 + R231;
double R233 = R176 + R177 + R178 + R179 + R180 + R181 + R182 + R183 + R184 + R185 + R186 + R187 + R188 + R189 + R190 + R191 + R192;
double R234 = R176 * R34;
double R235 = R177 * R35;
double R236 = R178 * R36;
double R237 = R179 * R37;
double R238 = R180 * R38;
double R239 = R181 * R39;
double R240 = R182 * R40;
double R241 = R183 * R41;
double R242 = R184 * R42;
double R243 = R185 * R43;
double R244 = R186 * R44;
double R245 = R187 * R45;
double R246 = R188 * R46;
double R247 = R189 * R47;
double R248 = R190 * R48;
double R249 = R191 * R49;
double R250 = R192 * R50;
double R251 = R234 + R235 + R236 + R237 + R238 + R239 + R240 + R241 + R242 + R243 + R244 + R245 + R246 + R247 + R248 + R249 + R250;
double R252 = R176 * R68;
double R253 = R177 * R69;
double R254 = R178 * R70;
double R255 = R179 * R71;
double R256 = R180 * R72;
double R257 = R181 * R73;
double R258 = R182 * R74;
double R259 = R183 * R75;
double R260 = R184 * R76;
double R261 = R185 * R77;
double R262 = R186 * R78;
double R263 = R187 * R79;
double R264 = R188 * R80;
double R265 = R189 * R81;
double R266 = R190 * R82;
double R267 = R191 * R83;
double R268 = R192 * R84;
double R269 = R252 + R253 + R254 + R255 + R256 + R257 + R258 + R259 + R260 + R261 + R262 + R263 + R264 + R265 + R266 + R267 + R268;
double R270 = R193 * R121 * R119;
double R271 = R120 * R158;
double R272 = R270 + R271;
double R273 = R85 * R68;
double R274 = R86 * R69;
double R275 = R87 * R70;
double R276 = R88 * R71;
double R277 = R89 * R72;
double R278 = R90 * R73;
double R279 = R91 * R74;
double R280 = R92 * R75;
double R281 = R93 * R76;
double R282 = R94 * R77;
double R283 = R95 * R78;
double R284 = R96 * R79;
double R285 = R97 * R80;
double R286 = R98 * R81;
double R287 = R99 * R82;
double R288 = R100 * R83;
double R289 = R101 * R84;
double R290 = R273 + R274 + R275 + R276 + R277 + R278 + R279 + R280 + R281 + R282 + R283 + R284 + R285 + R286 + R287 + R288 + R289;
double R291 = R193 * R233 * R119;
double R292 = R120 * R251;
double R293 = R291 + R292;
double R294 = R193 * R121 * R139;
double R295 = R140 * R158;
double R296 = R294 + R295;
double R297 = R193 * R233 * R139;
double R298 = R140 * R251;
double R299 = R297 + R298;
double R300 = R193 * R233 * R158;
double R301 = R121 * R251;
double R302 = R300 + R301;
double R303 = Power4( R34);
double R304 = R85 * R159;
double R305 = R86 * R160;
double R306 = R87 * R161;
double R307 = R88 * R162;
double R308 = R89 * R163;
double R309 = R90 * R164;
double R310 = R91 * R165;
double R311 = R92 * R166;
double R312 = R93 * R167;
double R313 = R94 * R168;
double R314 = R95 * R169;
double R315 = R96 * R170;
double R316 = R97 * R171;
double R317 = R98 * R172;
double R318 = R99 * R173;
double R319 = R100 * R174;
double R320 = R101 * R175;
double R321 = R304 + R305 + R306 + R307 + R308 + R309 + R310 + R311 + R312 + R313 + R314 + R315 + R316 + R317 + R318 + R319 + R320;
double R322 = R34 + R35 + R36 + R37 + R38 + R39 + R40 + R41 + R42 + R43 + R44 + R45 + R46 + R47 + R48 + R49 + R50;
double R323 = R51 * R159;
double R324 = R52 * R160;
double R325 = R53 * R161;
double R326 = R54 * R162;
double R327 = R55 * R163;
double R328 = R56 * R164;
double R329 = R57 * R165;
double R330 = R58 * R166;
double R331 = R59 * R167;
double R332 = R60 * R168;
double R333 = R61 * R169;
double R334 = R62 * R170;
double R335 = R63 * R171;
double R336 = R64 * R172;
double R337 = R65 * R173;
double R338 = R66 * R174;
double R339 = R67 * R175;
double R340 = R323 + R324 + R325 + R326 + R327 + R328 + R329 + R330 + R331 + R332 + R333 + R334 + R335 + R336 + R337 + R338 + R339;
double R341 = R193 * R120 * R322;
double R343 = R342 * R119;
double R344 = R341 + R343;
double R345 = R68 + R69 + R70 + R71 + R72 + R73 + R74 + R75 + R76 + R77 + R78 + R79 + R80 + R81 + R82 + R83 + R84;
double R346 = A[0] * R159;
double R347 = A[9] * R160;
double R348 = A[10] * R161;
double R349 = A[11] * R162;
double R350 = A[12] * R163;
double R351 = A[13] * R164;
double R352 = A[14] * R165;
double R353 = A[15] * R166;
double R354 = A[16] * R167;
double R355 = A[1] * R168;
double R356 = A[2] * R169;
double R357 = A[3] * R170;
double R358 = A[4] * R171;
double R359 = A[5] * R172;
double R360 = A[6] * R173;
double R361 = A[7] * R174;
double R362 = A[8] * R175;
double R363 = R346 + R347 + R348 + R349 + R350 + R351 + R352 + R353 + R354 + R355 + R356 + R357 + R358 + R359 + R360 + R361 + R362;
double R364 = R193 * R140 * R322;
double R365 = R342 * R139;
double R366 = R364 + R365;
double R367 = R193 * R121 * R322;
double R368 = R342 * R158;
double R369 = R367 + R368;
double R370 = R290 * R196;
double R371 = R193 * R214 * R272;
double R372 = R232 * R296;
double R373 = R370 + R371 + R372;
double R374 = Power4( R35);
double R375 = Power4( R36);
double R376 = Power4( R37);
double R377 = Power4( R38);
double R378 = Power4( R39);
double R379 = Power4( R40);
double R380 = Power4( R41);
double R381 = Power4( R42);
double R382 = Power4( R43);
double R383 = Power4( R44);
double R384 = Power4( R45);
double R385 = Power4( R46);
double R386 = Power4( R47);
double R387 = Power4( R48);
double R388 = Power4( R49);
double R389 = Power4( R50);
double R390 = R176 * R159;
double R391 = R177 * R160;
double R392 = R178 * R161;
double R393 = R179 * R162;
double R394 = R180 * R163;
double R395 = R181 * R164;
double R396 = R182 * R165;
double R397 = R183 * R166;
double R398 = R184 * R167;
double R399 = R185 * R168;
double R400 = R186 * R169;
double R401 = R187 * R170;
double R402 = R188 * R171;
double R403 = R189 * R172;
double R404 = R190 * R173;
double R405 = R191 * R174;
double R406 = R192 * R175;
double R407 = R390 + R391 + R392 + R393 + R394 + R395 + R396 + R397 + R398 + R399 + R400 + R401 + R402 + R403 + R404 + R405 + R406;
double R408 = R214 * R344;
double R409 = R193 * R232 * R366;
double R410 = R345 * R196;
double R411 = R408 + R409 + R410;
double R412 = R193 * R233 * R322;
double R413 = R342 * R251;
double R414 = R412 + R413;
double R415 = R159 + R160 + R161 + R162 + R163 + R164 + R165 + R166 + R167 + R168 + R169 + R170 + R171 + R172 + R173 + R174 + R175;
double R416 = R269 * R196;
double R417 = R193 * R214 * R293;
double R418 = R232 * R299;
double R419 = R416 + R417 + R418;
double R420 = R290 * R344;
double R421 = R193 * R232 * R369;
double R422 = R345 * R272;
double R423 = R420 + R421 + R422;
double R424 = R269 * R344;
double R425 = R193 * R232 * R414;
double R426 = R345 * R293;
double R427 = R424 + R425 + R426;
double R428 = R269 * R272;
double R429 = R193 * R290 * R293;
double R430 = R232 * R302;
double R431 = R428 + R429 + R430;
double R432 = R290 * R366;
double R433 = R193 * R214 * R369;
double R434 = R345 * R296;
double R435 = R432 + R433 + R434;
double R436 = R269 * R366;
double R437 = R193 * R214 * R414;
double R438 = R345 * R299;
double R439 = R436 + R437 + R438;
double R440 = R269 * R369;
double R441 = R193 * R290 * R414;
double R442 = R345 * R302;
double R443 = R440 + R441 + R442;
double R444 = R269 * R296;
double R445 = R193 * R290 * R299;
double R446 = R214 * R302;
double R447 = R444 + R445 + R446;
double R448 = R407 * R373;
double R449 = R193 * R321 * R419;
double R450 = R340 * R431;
double R451 = R193 * R363 * R447;
double R452 = R448 + R449 + R450 + R451;
double R453 = R176 * R303;
double R454 = R177 * R374;
double R455 = R178 * R375;
double R456 = R179 * R376;
double R457 = R180 * R377;
double R458 = R181 * R378;
double R459 = R182 * R379;
double R460 = R183 * R380;
double R461 = R184 * R381;
double R462 = R185 * R382;
double R463 = R186 * R383;
double R464 = R187 * R384;
double R465 = R188 * R385;
double R466 = R189 * R386;
double R467 = R190 * R387;
double R468 = R191 * R388;
double R469 = R192 * R389;
double R470 = R453 + R454 + R455 + R456 + R457 + R458 + R459 + R460 + R461 + R462 + R463 + R464 + R465 + R466 + R467 + R468 + R469;
double R471 = R85 * R303;
double R472 = R86 * R374;
double R473 = R87 * R375;
double R474 = R88 * R376;
double R475 = R89 * R377;
double R476 = R90 * R378;
double R477 = R91 * R379;
double R478 = R92 * R380;
double R479 = R93 * R381;
double R480 = R94 * R382;
double R481 = R95 * R383;
double R482 = R96 * R384;
double R483 = R97 * R385;
double R484 = R98 * R386;
double R485 = R99 * R387;
double R486 = R100 * R388;
double R487 = R101 * R389;
double R488 = R471 + R472 + R473 + R474 + R475 + R476 + R477 + R478 + R479 + R480 + R481 + R482 + R483 + R484 + R485 + R486 + R487;
double R489 = R51 * R303;
double R490 = R52 * R374;
double R491 = R53 * R375;
double R492 = R54 * R376;
double R493 = R55 * R377;
double R494 = R56 * R378;
double R495 = R57 * R379;
double R496 = R58 * R380;
double R497 = R59 * R381;
double R498 = R60 * R382;
double R499 = R61 * R383;
double R500 = R62 * R384;
double R501 = R63 * R385;
double R502 = R64 * R386;
double R503 = R65 * R387;
double R504 = R66 * R388;
double R505 = R67 * R389;
double R506 = R489 + R490 + R491 + R492 + R493 + R494 + R495 + R496 + R497 + R498 + R499 + R500 + R501 + R502 + R503 + R504 + R505;
double R507 = A[0] * R303;
double R508 = A[9] * R374;
double R509 = A[10] * R375;
double R510 = A[11] * R376;
double R511 = A[12] * R377;
double R512 = A[13] * R378;
double R513 = A[14] * R379;
double R514 = A[15] * R380;
double R515 = A[16] * R381;
double R516 = A[1] * R382;
double R517 = A[2] * R383;
double R518 = A[3] * R384;
double R519 = A[4] * R385;
double R520 = A[5] * R386;
double R521 = A[6] * R387;
double R522 = A[7] * R388;
double R523 = A[8] * R389;
double R524 = R507 + R508 + R509 + R510 + R511 + R512 + R513 + R514 + R515 + R516 + R517 + R518 + R519 + R520 + R521 + R522 + R523;
double R525 = R321 * R411;
double R526 = R193 * R340 * R423;
double R527 = R363 * R435;
double R528 = R193 * R415 * R373;
double R529 = R525 + R526 + R527 + R528;
double R530 = R470 * R529;
double R531 = R407 * R411;
double R532 = R193 * R340 * R427;
double R533 = R363 * R439;
double R534 = R193 * R415 * R419;
double R535 = R531 + R532 + R533 + R534;
double R536 = R193 * R488 * R535;
double R537 = R407 * R423;
double R538 = R193 * R321 * R427;
double R539 = R363 * R443;
double R540 = R193 * R415 * R431;
double R541 = R537 + R538 + R539 + R540;
double R542 = R506 * R541;
double R543 = R407 * R435;
double R544 = R193 * R321 * R439;
double R545 = R340 * R443;
double R546 = R193 * R415 * R447;
double R547 = R543 + R544 + R545 + R546;
double R548 = R193 * R524 * R547;
double R549 = R303 + R374 + R375 + R376 + R377 + R378 + R379 + R380 + R381 + R382 + R383 + R384 + R385 + R386 + R387 + R388 + R389;
double R550 = R549 * R452;
double R551 = R530 + R536 + R542 + R548 + R550;
double R552 = Reciprocal( R551);
double R553 = R193 * R140 * R232;
double R554 = R120 * R214;
double R555 = R553 + R554;
double R556 = R193 * R121 * R232;
double R557 = R120 * R290;
double R558 = R556 + R557;
double R559 = R193 * R233 * R232;
double R560 = R120 * R269;
double R561 = R559 + R560;
double R562 = R193 * R121 * R214;
double R563 = R140 * R290;
double R564 = R562 + R563;
double R565 = R193 * R233 * R214;
double R566 = R140 * R269;
double R567 = R565 + R566;
double R568 = R193 * R233 * R290;
double R569 = R121 * R269;
double R570 = R568 + R569;
double R571 = R193 * R139 * R232;
double R572 = R119 * R214;
double R573 = R571 + R572;
double R574 = R193 * R158 * R232;
double R575 = R119 * R290;
double R576 = R574 + R575;
double R577 = R193 * R251 * R232;
double R578 = R119 * R269;
double R579 = R577 + R578;
double R580 = R193 * R158 * R214;
double R581 = R139 * R290;
double R582 = R580 + R581;
double R583 = R193 * R251 * R214;
double R584 = R139 * R269;
double R585 = R583 + R584;
double R586 = R193 * R251 * R290;
double R587 = R158 * R269;
double R588 = R586 + R587;
double R589 = R193 * R470 * R373;
double R590 = R488 * R419;
double R591 = R193 * R506 * R431;
double R592 = R524 * R447;
double R593 = R589 + R590 + R591 + R592;
double R594 = R321 * R196;
double R595 = R193 * R340 * R272;
double R596 = R363 * R296;
double R597 = R594 + R595 + R596;
double R598 = R470 * R597;
double R599 = R407 * R196;
double R600 = R193 * R340 * R293;
double R601 = R363 * R299;
double R602 = R599 + R600 + R601;
double R603 = R193 * R488 * R602;
double R604 = R407 * R272;
double R605 = R193 * R321 * R293;
double R606 = R363 * R302;
double R607 = R604 + R605 + R606;
double R608 = R506 * R607;
double R609 = R407 * R296;
double R610 = R193 * R321 * R299;
double R611 = R340 * R302;
double R612 = R609 + R610 + R611;
double R613 = R193 * R524 * R612;
double R614 = R598 + R603 + R608 + R613;
double R615 = R321 * R555;
double R616 = R193 * R340 * R558;
double R617 = R363 * R564;
double R618 = R615 + R616 + R617;
double R619 = R193 * R470 * R618;
double R620 = R407 * R555;
double R621 = R193 * R340 * R561;
double R622 = R363 * R567;
double R623 = R620 + R621 + R622;
double R624 = R488 * R623;
double R625 = R407 * R558;
double R626 = R193 * R321 * R561;
double R627 = R363 * R570;
double R628 = R625 + R626 + R627;
double R629 = R193 * R506 * R628;
double R630 = R407 * R564;
double R631 = R193 * R321 * R567;
double R632 = R340 * R570;
double R633 = R630 + R631 + R632;
double R634 = R524 * R633;
double R635 = R619 + R624 + R629 + R634;
double R636 = R321 * R573;
double R637 = R193 * R340 * R576;
double R638 = R363 * R582;
double R639 = R636 + R637 + R638;
double R640 = R470 * R639;
double R641 = R407 * R573;
double R642 = R193 * R340 * R579;
double R643 = R363 * R585;
double R644 = R641 + R642 + R643;
double R645 = R193 * R488 * R644;
double R646 = R407 * R576;
double R647 = R193 * R321 * R579;
double R648 = R363 * R588;
double R649 = R646 + R647 + R648;
double R650 = R506 * R649;
double R651 = R407 * R582;
double R652 = R193 * R321 * R585;
double R653 = R340 * R588;
double R654 = R651 + R652 + R653;
double R655 = R193 * R524 * R654;
double R656 = R640 + R645 + R650 + R655;
double R657 = R656 * R552;
double R658 = R193 * R140 * R345;
double R659 = R342 * R214;
double R660 = R658 + R659;
double R661 = R193 * R121 * R345;
double R662 = R342 * R290;
double R663 = R661 + R662;
double R664 = R193 * R233 * R345;
double R665 = R342 * R269;
double R666 = R664 + R665;
double R667 = R193 * R139 * R345;
double R668 = R322 * R214;
double R669 = R667 + R668;
double R670 = R193 * R158 * R345;
double R671 = R322 * R290;
double R672 = R670 + R671;
double R673 = R193 * R251 * R345;
double R674 = R322 * R269;
double R675 = R673 + R674;
double R676 = R193 * R407 * R435;
double R677 = R321 * R439;
double R678 = R193 * R340 * R443;
double R679 = R415 * R447;
double R680 = R676 + R677 + R678 + R679;
double R681 = R470 * R435;
double R682 = R193 * R488 * R439;
double R683 = R506 * R443;
double R684 = R193 * R549 * R447;
double R685 = R681 + R682 + R683 + R684;
double R686 = R321 * R366;
double R687 = R193 * R340 * R369;
double R688 = R415 * R296;
double R689 = R686 + R687 + R688;
double R690 = R193 * R470 * R689;
double R691 = R407 * R366;
double R692 = R193 * R340 * R414;
double R693 = R415 * R299;
double R694 = R691 + R692 + R693;
double R695 = R488 * R694;
double R696 = R407 * R369;
double R697 = R193 * R321 * R414;
double R698 = R415 * R302;
double R699 = R696 + R697 + R698;
double R700 = R193 * R506 * R699;
double R701 = R549 * R612;
double R702 = R690 + R695 + R700 + R701;
double R703 = R321 * R660;
double R704 = R193 * R340 * R663;
double R705 = R415 * R564;
double R706 = R703 + R704 + R705;
double R707 = R470 * R706;
double R708 = R407 * R660;
double R709 = R193 * R340 * R666;
double R710 = R415 * R567;
double R711 = R708 + R709 + R710;
double R712 = R193 * R488 * R711;
double R713 = R407 * R663;
double R714 = R193 * R321 * R666;
double R715 = R415 * R570;
double R716 = R713 + R714 + R715;
double R717 = R506 * R716;
double R718 = R193 * R549 * R633;
double R719 = R707 + R712 + R717 + R718;
double R720 = R321 * R669;
double R721 = R193 * R340 * R672;
double R722 = R415 * R582;
double R723 = R720 + R721 + R722;
double R724 = R193 * R470 * R723;
double R725 = R407 * R669;
double R726 = R193 * R340 * R675;
double R727 = R415 * R585;
double R728 = R725 + R726 + R727;
double R729 = R488 * R728;
double R730 = R407 * R672;
double R731 = R193 * R321 * R675;
double R732 = R415 * R588;
double R733 = R730 + R731 + R732;
double R734 = R193 * R506 * R733;
double R735 = R549 * R654;
double R736 = R724 + R729 + R734 + R735;
double R737 = R736 * R552;
double R738 = R193 * R120 * R345;
double R739 = R342 * R232;
double R740 = R738 + R739;
double R741 = R193 * R119 * R345;
double R742 = R322 * R232;
double R743 = R741 + R742;
double R744 = R193 * R470 * R423;
double R745 = R488 * R427;
double R746 = R193 * R524 * R443;
double R747 = R549 * R431;
double R748 = R744 + R745 + R746 + R747;
double R749 = R321 * R344;
double R750 = R193 * R363 * R369;
double R751 = R415 * R272;
double R752 = R749 + R750 + R751;
double R753 = R470 * R752;
double R754 = R407 * R344;
double R755 = R193 * R363 * R414;
double R756 = R415 * R293;
double R757 = R754 + R755 + R756;
double R758 = R193 * R488 * R757;
double R759 = R524 * R699;
double R760 = R193 * R549 * R607;
double R761 = R753 + R758 + R759 + R760;
double R762 = R321 * R740;
double R763 = R193 * R363 * R663;
double R764 = R415 * R558;
double R765 = R762 + R763 + R764;
double R766 = R193 * R470 * R765;
double R767 = R407 * R740;
double R768 = R193 * R363 * R666;
double R769 = R415 * R561;
double R770 = R767 + R768 + R769;
double R771 = R488 * R770;
double R772 = R193 * R524 * R716;
double R773 = R549 * R628;
double R774 = R766 + R771 + R772 + R773;
double R775 = R321 * R743;
double R776 = R193 * R363 * R672;
double R777 = R415 * R576;
double R778 = R775 + R776 + R777;
double R779 = R470 * R778;
double R780 = R407 * R743;
double R781 = R193 * R363 * R675;
double R782 = R415 * R579;
double R783 = R780 + R781 + R782;
double R784 = R193 * R488 * R783;
double R785 = R524 * R733;
double R786 = R193 * R549 * R649;
double R787 = R779 + R784 + R785 + R786;
double R788 = R787 * R552;
double R789 = R193 * R407 * R411;
double R790 = R340 * R427;
double R791 = R193 * R363 * R439;
double R792 = R415 * R419;
double R793 = R789 + R790 + R791 + R792;
double R794 = R470 * R411;
double R795 = R193 * R506 * R427;
double R796 = R524 * R439;
double R797 = R193 * R549 * R419;
double R798 = R794 + R795 + R796 + R797;
double R799 = R340 * R344;
double R800 = R193 * R363 * R366;
double R801 = R415 * R196;
double R802 = R799 + R800 + R801;
double R803 = R193 * R470 * R802;
double R804 = R506 * R757;
double R805 = R193 * R524 * R694;
double R806 = R549 * R602;
double R807 = R803 + R804 + R805 + R806;
double R808 = R340 * R740;
double R809 = R193 * R363 * R660;
double R810 = R415 * R555;
double R811 = R808 + R809 + R810;
double R812 = R470 * R811;
double R813 = R193 * R506 * R770;
double R814 = R524 * R711;
double R815 = R193 * R549 * R623;
double R816 = R812 + R813 + R814 + R815;
double R817 = R340 * R743;
double R818 = R193 * R363 * R669;
double R819 = R415 * R573;
double R820 = R817 + R818 + R819;
double R821 = R193 * R470 * R820;
double R822 = R506 * R783;
double R823 = R193 * R524 * R728;
double R824 = R549 * R644;
double R825 = R821 + R822 + R823 + R824;
double R826 = R825 * R552;
double R827 = R193 * R488 * R411;
double R828 = R506 * R423;
double R829 = R193 * R524 * R435;
double R830 = R549 * R373;
double R831 = R827 + R828 + R829 + R830;
double R832 = R488 * R802;
double R833 = R193 * R506 * R752;
double R834 = R524 * R689;
double R835 = R193 * R549 * R597;
double R836 = R832 + R833 + R834 + R835;
double R837 = R193 * R488 * R811;
double R838 = R506 * R765;
double R839 = R193 * R524 * R706;
double R840 = R549 * R618;
double R841 = R837 + R838 + R839 + R840;
double R842 = R488 * R820;
double R843 = R193 * R506 * R778;
double R844 = R524 * R723;
double R845 = R193 * R549 * R639;
double R846 = R842 + R843 + R844 + R845;
double R847 = R846 * R552;
double R848 = R303 * R452 * R552;
double R849 = R159 * R593 * R552;
double R850 = R68 * R614 * R552;
double R851 = R34 * R635 * R552;
R848 = R848 + R849 + R850 + R851 + R657;
R849 = A[17] * R848;
R848 = R374 * R452 * R552;
R850 = R160 * R593 * R552;
R851 = R69 * R614 * R552;
double R852 = R35 * R635 * R552;
R848 = R848 + R850 + R851 + R852 + R657;
R850 = A[26] * R848;
R848 = R375 * R452 * R552;
R851 = R161 * R593 * R552;
R852 = R70 * R614 * R552;
double R853 = R36 * R635 * R552;
R848 = R848 + R851 + R852 + R853 + R657;
R851 = A[27] * R848;
R848 = R376 * R452 * R552;
R852 = R162 * R593 * R552;
R853 = R71 * R614 * R552;
double R854 = R37 * R635 * R552;
R848 = R848 + R852 + R853 + R854 + R657;
R852 = A[28] * R848;
R848 = R377 * R452 * R552;
R853 = R163 * R593 * R552;
R854 = R72 * R614 * R552;
double R855 = R38 * R635 * R552;
R848 = R848 + R853 + R854 + R855 + R657;
R853 = A[29] * R848;
R848 = R378 * R452 * R552;
R854 = R164 * R593 * R552;
R855 = R73 * R614 * R552;
double R856 = R39 * R635 * R552;
R848 = R848 + R854 + R855 + R856 + R657;
R854 = A[30] * R848;
R848 = R379 * R452 * R552;
R855 = R165 * R593 * R552;
R856 = R74 * R614 * R552;
double R857 = R40 * R635 * R552;
R848 = R848 + R855 + R856 + R857 + R657;
R855 = A[31] * R848;
R848 = R380 * R452 * R552;
R856 = R166 * R593 * R552;
R857 = R75 * R614 * R552;
double R858 = R41 * R635 * R552;
R848 = R848 + R856 + R857 + R858 + R657;
R856 = A[32] * R848;
R848 = R381 * R452 * R552;
R857 = R167 * R593 * R552;
R858 = R76 * R614 * R552;
double R859 = R42 * R635 * R552;
R848 = R848 + R857 + R858 + R859 + R657;
R857 = A[33] * R848;
R848 = R382 * R452 * R552;
R858 = R168 * R593 * R552;
R859 = R77 * R614 * R552;
double R860 = R43 * R635 * R552;
R848 = R848 + R858 + R859 + R860 + R657;
R858 = A[18] * R848;
R848 = R383 * R452 * R552;
R859 = R169 * R593 * R552;
R860 = R78 * R614 * R552;
double R861 = R44 * R635 * R552;
R848 = R848 + R859 + R860 + R861 + R657;
R859 = A[19] * R848;
R848 = R384 * R452 * R552;
R860 = R170 * R593 * R552;
R861 = R79 * R614 * R552;
double R862 = R45 * R635 * R552;
R848 = R848 + R860 + R861 + R862 + R657;
R860 = A[20] * R848;
R848 = R385 * R452 * R552;
R861 = R171 * R593 * R552;
R862 = R80 * R614 * R552;
double R863 = R46 * R635 * R552;
R848 = R848 + R861 + R862 + R863 + R657;
R861 = A[21] * R848;
R848 = R386 * R452 * R552;
R862 = R172 * R593 * R552;
R863 = R81 * R614 * R552;
double R864 = R47 * R635 * R552;
R848 = R848 + R862 + R863 + R864 + R657;
R862 = A[22] * R848;
R848 = R387 * R452 * R552;
R863 = R173 * R593 * R552;
R864 = R82 * R614 * R552;
double R865 = R48 * R635 * R552;
R848 = R848 + R863 + R864 + R865 + R657;
R863 = A[23] * R848;
R848 = R388 * R452 * R552;
R864 = R174 * R593 * R552;
R865 = R83 * R614 * R552;
double R866 = R49 * R635 * R552;
R848 = R848 + R864 + R865 + R866 + R657;
R864 = A[24] * R848;
R848 = R389 * R452 * R552;
R865 = R175 * R593 * R552;
R866 = R84 * R614 * R552;
double R867 = R50 * R635 * R552;
R848 = R848 + R865 + R866 + R867 + R657;
R865 = A[25] * R848;
R849 = R849 + R850 + R851 + R852 + R853 + R854 + R855 + R856 + R857 + R858 + R859 + R860 + R861 + R862 + R863 + R864 + R865;
R850 = R303 * R680 * R552;
R851 = R159 * R685 * R552;
R852 = R68 * R702 * R552;
R853 = R34 * R719 * R552;
R850 = R850 + R851 + R852 + R853 + R737;
R851 = A[17] * R850;
R850 = R374 * R680 * R552;
R852 = R160 * R685 * R552;
R853 = R69 * R702 * R552;
R854 = R35 * R719 * R552;
R850 = R850 + R852 + R853 + R854 + R737;
R852 = A[26] * R850;
R850 = R375 * R680 * R552;
R853 = R161 * R685 * R552;
R854 = R70 * R702 * R552;
R855 = R36 * R719 * R552;
R850 = R850 + R853 + R854 + R855 + R737;
R853 = A[27] * R850;
R850 = R376 * R680 * R552;
R854 = R162 * R685 * R552;
R855 = R71 * R702 * R552;
R856 = R37 * R719 * R552;
R850 = R850 + R854 + R855 + R856 + R737;
R854 = A[28] * R850;
R850 = R377 * R680 * R552;
R855 = R163 * R685 * R552;
R856 = R72 * R702 * R552;
R857 = R38 * R719 * R552;
R850 = R850 + R855 + R856 + R857 + R737;
R855 = A[29] * R850;
R850 = R378 * R680 * R552;
R856 = R164 * R685 * R552;
R857 = R73 * R702 * R552;
R858 = R39 * R719 * R552;
R850 = R850 + R856 + R857 + R858 + R737;
R856 = A[30] * R850;
R850 = R379 * R680 * R552;
R857 = R165 * R685 * R552;
R858 = R74 * R702 * R552;
R859 = R40 * R719 * R552;
R850 = R850 + R857 + R858 + R859 + R737;
R857 = A[31] * R850;
R850 = R380 * R680 * R552;
R858 = R166 * R685 * R552;
R859 = R75 * R702 * R552;
R860 = R41 * R719 * R552;
R850 = R850 + R858 + R859 + R860 + R737;
R858 = A[32] * R850;
R850 = R381 * R680 * R552;
R859 = R167 * R685 * R552;
R860 = R76 * R702 * R552;
R861 = R42 * R719 * R552;
R850 = R850 + R859 + R860 + R861 + R737;
R859 = A[33] * R850;
R850 = R382 * R680 * R552;
R860 = R168 * R685 * R552;
R861 = R77 * R702 * R552;
R862 = R43 * R719 * R552;
R850 = R850 + R860 + R861 + R862 + R737;
R860 = A[18] * R850;
R850 = R383 * R680 * R552;
R861 = R169 * R685 * R552;
R862 = R78 * R702 * R552;
R863 = R44 * R719 * R552;
R850 = R850 + R861 + R862 + R863 + R737;
R861 = A[19] * R850;
R850 = R384 * R680 * R552;
R862 = R170 * R685 * R552;
R863 = R79 * R702 * R552;
R864 = R45 * R719 * R552;
R850 = R850 + R862 + R863 + R864 + R737;
R862 = A[20] * R850;
R850 = R385 * R680 * R552;
R863 = R171 * R685 * R552;
R864 = R80 * R702 * R552;
R865 = R46 * R719 * R552;
R850 = R850 + R863 + R864 + R865 + R737;
R863 = A[21] * R850;
R850 = R386 * R680 * R552;
R864 = R172 * R685 * R552;
R865 = R81 * R702 * R552;
R848 = R47 * R719 * R552;
R850 = R850 + R864 + R865 + R848 + R737;
R864 = A[22] * R850;
R850 = R387 * R680 * R552;
R865 = R173 * R685 * R552;
R848 = R82 * R702 * R552;
R866 = R48 * R719 * R552;
R850 = R850 + R865 + R848 + R866 + R737;
R865 = A[23] * R850;
R850 = R388 * R680 * R552;
R848 = R174 * R685 * R552;
R866 = R83 * R702 * R552;
R867 = R49 * R719 * R552;
R850 = R850 + R848 + R866 + R867 + R737;
R848 = A[24] * R850;
R850 = R389 * R680 * R552;
R866 = R175 * R685 * R552;
R867 = R84 * R702 * R552;
double R868 = R50 * R719 * R552;
R850 = R850 + R866 + R867 + R868 + R737;
R866 = A[25] * R850;
R851 = R851 + R852 + R853 + R854 + R855 + R856 + R857 + R858 + R859 + R860 + R861 + R862 + R863 + R864 + R865 + R848 + R866;
R852 = R303 * R541 * R552;
R853 = R159 * R748 * R552;
R854 = R68 * R761 * R552;
R855 = R34 * R774 * R552;
R852 = R852 + R853 + R854 + R855 + R788;
R853 = A[17] * R852;
R852 = R374 * R541 * R552;
R854 = R160 * R748 * R552;
R855 = R69 * R761 * R552;
R856 = R35 * R774 * R552;
R852 = R852 + R854 + R855 + R856 + R788;
R854 = A[26] * R852;
R852 = R375 * R541 * R552;
R855 = R161 * R748 * R552;
R856 = R70 * R761 * R552;
R857 = R36 * R774 * R552;
R852 = R852 + R855 + R856 + R857 + R788;
R855 = A[27] * R852;
R852 = R376 * R541 * R552;
R856 = R162 * R748 * R552;
R857 = R71 * R761 * R552;
R858 = R37 * R774 * R552;
R852 = R852 + R856 + R857 + R858 + R788;
R856 = A[28] * R852;
R852 = R377 * R541 * R552;
R857 = R163 * R748 * R552;
R858 = R72 * R761 * R552;
R859 = R38 * R774 * R552;
R852 = R852 + R857 + R858 + R859 + R788;
R857 = A[29] * R852;
R852 = R378 * R541 * R552;
R858 = R164 * R748 * R552;
R859 = R73 * R761 * R552;
R860 = R39 * R774 * R552;
R852 = R852 + R858 + R859 + R860 + R788;
R858 = A[30] * R852;
R852 = R379 * R541 * R552;
R859 = R165 * R748 * R552;
R860 = R74 * R761 * R552;
R861 = R40 * R774 * R552;
R852 = R852 + R859 + R860 + R861 + R788;
R859 = A[31] * R852;
R852 = R380 * R541 * R552;
R860 = R166 * R748 * R552;
R861 = R75 * R761 * R552;
R862 = R41 * R774 * R552;
R852 = R852 + R860 + R861 + R862 + R788;
R860 = A[32] * R852;
R852 = R381 * R541 * R552;
R861 = R167 * R748 * R552;
R862 = R76 * R761 * R552;
R863 = R42 * R774 * R552;
R852 = R852 + R861 + R862 + R863 + R788;
R861 = A[33] * R852;
R852 = R382 * R541 * R552;
R862 = R168 * R748 * R552;
R863 = R77 * R761 * R552;
R864 = R43 * R774 * R552;
R852 = R852 + R862 + R863 + R864 + R788;
R862 = A[18] * R852;
R852 = R383 * R541 * R552;
R863 = R169 * R748 * R552;
R864 = R78 * R761 * R552;
R865 = R44 * R774 * R552;
R852 = R852 + R863 + R864 + R865 + R788;
R863 = A[19] * R852;
R852 = R384 * R541 * R552;
R864 = R170 * R748 * R552;
R865 = R79 * R761 * R552;
R848 = R45 * R774 * R552;
R852 = R852 + R864 + R865 + R848 + R788;
R864 = A[20] * R852;
R852 = R385 * R541 * R552;
R865 = R171 * R748 * R552;
R848 = R80 * R761 * R552;
R866 = R46 * R774 * R552;
R852 = R852 + R865 + R848 + R866 + R788;
R865 = A[21] * R852;
R852 = R386 * R541 * R552;
R848 = R172 * R748 * R552;
R866 = R81 * R761 * R552;
R850 = R47 * R774 * R552;
R852 = R852 + R848 + R866 + R850 + R788;
R848 = A[22] * R852;
R852 = R387 * R541 * R552;
R866 = R173 * R748 * R552;
R850 = R82 * R761 * R552;
R867 = R48 * R774 * R552;
R852 = R852 + R866 + R850 + R867 + R788;
R866 = A[23] * R852;
R852 = R388 * R541 * R552;
R850 = R174 * R748 * R552;
R867 = R83 * R761 * R552;
R868 = R49 * R774 * R552;
R852 = R852 + R850 + R867 + R868 + R788;
R850 = A[24] * R852;
R852 = R389 * R541 * R552;
R867 = R175 * R748 * R552;
R868 = R84 * R761 * R552;
double R869 = R50 * R774 * R552;
R852 = R852 + R867 + R868 + R869 + R788;
R867 = A[25] * R852;
R853 = R853 + R854 + R855 + R856 + R857 + R858 + R859 + R860 + R861 + R862 + R863 + R864 + R865 + R848 + R866 + R850 + R867;
R854 = R303 * R793 * R552;
R855 = R159 * R798 * R552;
R856 = R68 * R807 * R552;
R857 = R34 * R816 * R552;
R854 = R854 + R855 + R856 + R857 + R826;
R855 = A[17] * R854;
R854 = R374 * R793 * R552;
R856 = R160 * R798 * R552;
R857 = R69 * R807 * R552;
R858 = R35 * R816 * R552;
R854 = R854 + R856 + R857 + R858 + R826;
R856 = A[26] * R854;
R854 = R375 * R793 * R552;
R857 = R161 * R798 * R552;
R858 = R70 * R807 * R552;
R859 = R36 * R816 * R552;
R854 = R854 + R857 + R858 + R859 + R826;
R857 = A[27] * R854;
R854 = R376 * R793 * R552;
R858 = R162 * R798 * R552;
R859 = R71 * R807 * R552;
R860 = R37 * R816 * R552;
R854 = R854 + R858 + R859 + R860 + R826;
R858 = A[28] * R854;
R854 = R377 * R793 * R552;
R859 = R163 * R798 * R552;
R860 = R72 * R807 * R552;
R861 = R38 * R816 * R552;
R854 = R854 + R859 + R860 + R861 + R826;
R859 = A[29] * R854;
R854 = R378 * R793 * R552;
R860 = R164 * R798 * R552;
R861 = R73 * R807 * R552;
R862 = R39 * R816 * R552;
R854 = R854 + R860 + R861 + R862 + R826;
R860 = A[30] * R854;
R854 = R379 * R793 * R552;
R861 = R165 * R798 * R552;
R862 = R74 * R807 * R552;
R863 = R40 * R816 * R552;
R854 = R854 + R861 + R862 + R863 + R826;
R861 = A[31] * R854;
R854 = R380 * R793 * R552;
R862 = R166 * R798 * R552;
R863 = R75 * R807 * R552;
R864 = R41 * R816 * R552;
R854 = R854 + R862 + R863 + R864 + R826;
R862 = A[32] * R854;
R854 = R381 * R793 * R552;
R863 = R167 * R798 * R552;
R864 = R76 * R807 * R552;
R865 = R42 * R816 * R552;
R854 = R854 + R863 + R864 + R865 + R826;
R863 = A[33] * R854;
R854 = R382 * R793 * R552;
R864 = R168 * R798 * R552;
R865 = R77 * R807 * R552;
R848 = R43 * R816 * R552;
R854 = R854 + R864 + R865 + R848 + R826;
R864 = A[18] * R854;
R854 = R383 * R793 * R552;
R865 = R169 * R798 * R552;
R848 = R78 * R807 * R552;
R866 = R44 * R816 * R552;
R854 = R854 + R865 + R848 + R866 + R826;
R865 = A[19] * R854;
R854 = R384 * R793 * R552;
R848 = R170 * R798 * R552;
R866 = R79 * R807 * R552;
R850 = R45 * R816 * R552;
R854 = R854 + R848 + R866 + R850 + R826;
R848 = A[20] * R854;
R854 = R385 * R793 * R552;
R866 = R171 * R798 * R552;
R850 = R80 * R807 * R552;
R867 = R46 * R816 * R552;
R854 = R854 + R866 + R850 + R867 + R826;
R866 = A[21] * R854;
R854 = R386 * R793 * R552;
R850 = R172 * R798 * R552;
R867 = R81 * R807 * R552;
R852 = R47 * R816 * R552;
R854 = R854 + R850 + R867 + R852 + R826;
R850 = A[22] * R854;
R854 = R387 * R793 * R552;
R867 = R173 * R798 * R552;
R852 = R82 * R807 * R552;
R868 = R48 * R816 * R552;
R854 = R854 + R867 + R852 + R868 + R826;
R867 = A[23] * R854;
R854 = R388 * R793 * R552;
R852 = R174 * R798 * R552;
R868 = R83 * R807 * R552;
R869 = R49 * R816 * R552;
R854 = R854 + R852 + R868 + R869 + R826;
R852 = A[24] * R854;
R854 = R389 * R793 * R552;
R868 = R175 * R798 * R552;
R869 = R84 * R807 * R552;
double R870 = R50 * R816 * R552;
R854 = R854 + R868 + R869 + R870 + R826;
R868 = A[25] * R854;
R855 = R855 + R856 + R857 + R858 + R859 + R860 + R861 + R862 + R863 + R864 + R865 + R848 + R866 + R850 + R867 + R852 + R868;
R856 = R303 * R529 * R552;
R857 = R159 * R831 * R552;
R858 = R68 * R836 * R552;
R859 = R34 * R841 * R552;
R856 = R856 + R857 + R858 + R859 + R847;
R857 = A[17] * R856;
R856 = R374 * R529 * R552;
R858 = R160 * R831 * R552;
R859 = R69 * R836 * R552;
R860 = R35 * R841 * R552;
R856 = R856 + R858 + R859 + R860 + R847;
R858 = A[26] * R856;
R856 = R375 * R529 * R552;
R859 = R161 * R831 * R552;
R860 = R70 * R836 * R552;
R861 = R36 * R841 * R552;
R856 = R856 + R859 + R860 + R861 + R847;
R859 = A[27] * R856;
R856 = R376 * R529 * R552;
R860 = R162 * R831 * R552;
R861 = R71 * R836 * R552;
R862 = R37 * R841 * R552;
R856 = R856 + R860 + R861 + R862 + R847;
R860 = A[28] * R856;
R856 = R377 * R529 * R552;
R861 = R163 * R831 * R552;
R862 = R72 * R836 * R552;
R863 = R38 * R841 * R552;
R856 = R856 + R861 + R862 + R863 + R847;
R861 = A[29] * R856;
R856 = R378 * R529 * R552;
R862 = R164 * R831 * R552;
R863 = R73 * R836 * R552;
R864 = R39 * R841 * R552;
R856 = R856 + R862 + R863 + R864 + R847;
R862 = A[30] * R856;
R856 = R379 * R529 * R552;
R863 = R165 * R831 * R552;
R864 = R74 * R836 * R552;
R865 = R40 * R841 * R552;
R856 = R856 + R863 + R864 + R865 + R847;
R863 = A[31] * R856;
R856 = R380 * R529 * R552;
R864 = R166 * R831 * R552;
R865 = R75 * R836 * R552;
R848 = R41 * R841 * R552;
R856 = R856 + R864 + R865 + R848 + R847;
R864 = A[32] * R856;
R856 = R381 * R529 * R552;
R865 = R167 * R831 * R552;
R848 = R76 * R836 * R552;
R866 = R42 * R841 * R552;
R856 = R856 + R865 + R848 + R866 + R847;
R865 = A[33] * R856;
R856 = R382 * R529 * R552;
R848 = R168 * R831 * R552;
R866 = R77 * R836 * R552;
R850 = R43 * R841 * R552;
R856 = R856 + R848 + R866 + R850 + R847;
R848 = A[18] * R856;
R856 = R383 * R529 * R552;
R866 = R169 * R831 * R552;
R850 = R78 * R836 * R552;
R867 = R44 * R841 * R552;
R856 = R856 + R866 + R850 + R867 + R847;
R866 = A[19] * R856;
R856 = R384 * R529 * R552;
R850 = R170 * R831 * R552;
R867 = R79 * R836 * R552;
R852 = R45 * R841 * R552;
R856 = R856 + R850 + R867 + R852 + R847;
R850 = A[20] * R856;
R856 = R385 * R529 * R552;
R867 = R171 * R831 * R552;
R852 = R80 * R836 * R552;
R868 = R46 * R841 * R552;
R856 = R856 + R867 + R852 + R868 + R847;
R867 = A[21] * R856;
R856 = R386 * R529 * R552;
R852 = R172 * R831 * R552;
R868 = R81 * R836 * R552;
R854 = R47 * R841 * R552;
R856 = R856 + R852 + R868 + R854 + R847;
R852 = A[22] * R856;
R856 = R387 * R529 * R552;
R868 = R173 * R831 * R552;
R854 = R82 * R836 * R552;
R869 = R48 * R841 * R552;
R856 = R856 + R868 + R854 + R869 + R847;
R868 = A[23] * R856;
R856 = R388 * R529 * R552;
R854 = R174 * R831 * R552;
R869 = R83 * R836 * R552;
R870 = R49 * R841 * R552;
R856 = R856 + R854 + R869 + R870 + R847;
R854 = A[24] * R856;
R856 = R389 * R529 * R552;
R869 = R175 * R831 * R552;
R870 = R84 * R836 * R552;
double R871 = R50 * R841 * R552;
R856 = R856 + R869 + R870 + R871 + R847;
R869 = A[25] * R856;
R857 = R857 + R858 + R859 + R860 + R861 + R862 + R863 + R864 + R865 + R848 + R866 + R850 + R867 + R852 + R868 + R854 + R869;
output[0]=R849;
output[1]=R851;
output[2]=R853;
output[3]=R855;
output[4]=R857;
//T(R1)0 ={ R849, R851, R853, R855, R857 };
//Return;
}

__device__ __host__ inline double Square(double &x)
{
    return x*x;
}
__device__ __host__ inline double Reciprocal( double &x)
{
    return 1./x;
}
__device__ __host__ inline double Power3(double &x)
{
    return x*x*x;
}

__device__ __host__ inline double Power4(double &x)
{
    return x*x*x*x;
}
